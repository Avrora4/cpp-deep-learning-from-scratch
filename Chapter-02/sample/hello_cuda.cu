
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void add(int *a, int *b, int *c)
{
    *c = *a + *b;
}

int main()
{
    int a = 2;
    int b = 7;
    int c;
    int *d_a, *d_b, *d_c;

    // Allocate device memory
    hipMalloc((void**)&d_a, sizeof(int));
    hipMalloc((void**)&d_b, sizeof(int));
    hipMalloc((void**)&d_c, sizeof(int));

    // Copy inputs to device
    hipMemcpy(d_a, &a, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, &b, sizeof(int), hipMemcpyHostToDevice);
    
    // Launch kernel
    add<<<1,1>>>(&a,&b,&c);

    // Copy result back to host
    hipMemcpy(&c, d_c, sizeof(int), hipMemcpyDeviceToHost);

    printf("Result: %d\n", c);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}