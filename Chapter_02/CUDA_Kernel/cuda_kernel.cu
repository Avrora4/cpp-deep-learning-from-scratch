
// Kernel denition
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define CUDA_CHECK(call)\
    do\
    {\
        hipError_t err = (call);\
        if(err != hipSuccess)\
        {\
            fprintf(stderr, "CUDA error %s:%d: %s\n", __FILE__, __LINE__,hipGetErrorString(err));\
            exit(EXIT_FAILURE);\
        }\
    } while (0)\

__global__ void MatAdd(const float *A, const float *B, float *C, int N)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if(row< N && col < N)
    {
        int idx = row * N + col;
        C[idx] = A[idx] + B[idx];
    }
    // C[i][j] = A[i][j] + B[i][j];
}

int main()
{
    // float *A, *B, *C;
    int N = 16;
    size_t bytes = (size_t)N * N * sizeof(float);

    float *hA = (float*)malloc(bytes);
    float *hB = (float*)malloc(bytes);
    float *hC = (float*)malloc(bytes);
    if(!hA || !hB || !hC)
    {
        fprintf(stderr, "Failed to allocate host vectors!\n");
        exit(EXIT_FAILURE);
    }
    for (int i = 0; i < N * N; ++i)
    {
        hA[i] = 1.0f * i;
        hB[i] = 2.0f * i;
    }

    // Allocate memory on device
    /*
    cudaMalloc((void**)&A, N*N*sizeof(float));
    cudaMalloc((void**)&B, N*N*sizeof(float));
    cudaMalloc((void**)&C, N*N*sizeof(float));
    float *a = malloc(N*N*sizeof(float));
    float *b = malloc(N*N*sizeof(float));
    float *c = malloc(N*N*sizeof(float));
    */

    float *dA = NULL;
    float *dB = NULL;
    float *dC = NULL;
    CUDA_CHECK(hipMalloc((void**)&dA, bytes));
    CUDA_CHECK(hipMalloc((void**)&dB, bytes));
    CUDA_CHECK(hipMalloc((void**)&dC, bytes));

    CUDA_CHECK(hipMemcpy(dA, hA, bytes, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(dB, hB, bytes, hipMemcpyHostToDevice));


    // cudaMemcpy(A, a, N * N * sizeof(*A), cudaMemcpyHostToDevice);
    // cudaMemcpy(B, b, N * N * sizeof(*B), cudaMemcpyHostToDevice);

    // kernel invocation with one block of N * N * 1 threads
    // int numBlocks = 1;
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((N + threadsPerBlock.x - 1) / threadsPerBlock.x, 
                   (N + threadsPerBlock.y - 1) / threadsPerBlock.y);
    MatAdd<<<numBlocks, threadsPerBlock>>>(dA, dB, dC, N);

    // runtime error check
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    // Device -> Host copy
    CUDA_CHECK(hipMemcpy(hC, dC, bytes, hipMemcpyDeviceToHost));

    // results (Header 4 * 4)
    for (int r = 0; r < 4 && r < N; ++r)
    {
        for (int c = 0; c < 4 && c < N; ++c)
        {
            printf("%8.1f ", hC[r * N + c]);
        }
        printf("\n");
    }

    // cudaMecpy(c, C, N * N * sizeof(*c), cudaMemcpyDeviceToHost);

    // free device memory
    CUDA_CHECK(hipFree(dA));
    CUDA_CHECK(hipFree(dB));
    CUDA_CHECK(hipFree(dC));
    free(hA);
    free(hB);
    free(hC);

    return 0;

}